#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <hip/hip_fp16.h>
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/Dispatch.h>
#include <ATen/TensorUtils.h>
#include <ATen/TensorOperators.h>
#include <ATen/WrapDimUtils.h>
#include <c10/macros/Macros.h>

#include <ATen/AccumulateType.h>
#include <ATen/cuda/NumericLimits.cuh>
#include <type_traits>

#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/MemoryAccess.cuh>
#include <ATen/native/cuda/PersistentSoftmax.cuh>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/_masked_softmax_native.h>
#include <ATen/ops/_log_softmax_native.h>
#include <ATen/ops/_log_softmax_backward_data_native.h>
#include <ATen/ops/_softmax_native.h>
#include <ATen/ops/_softmax_backward_data_native.h>
#include <ATen/ops/softmax.h>
#include <ATen/ops/_softmax_backward_data.h>
#endif

namespace at {
namespace extend {

constexpr int ALIGN_BYTES = 16;
const int max_threads = 1024;

template<typename T, typename AccumT, typename OutT>
struct SoftMaxForwardEpilogue {
  __device__ __forceinline__ SoftMaxForwardEpilogue(AccumT max_input, AccumT sum, float bias)
    : max_input(max_input)
    , sum(sum)
    , bias(bias) {}

  __device__ __forceinline__ OutT operator()(T input) const {
    return static_cast<OutT>(std::exp(input - max_input) / sum + bias);
  }

  const AccumT max_input;
  const AccumT sum;
  const float bias;
};


template<>
struct SoftMaxForwardEpilogue<__half, float, __half> {
  __device__ __forceinline__ SoftMaxForwardEpilogue(float max_input, float sum, float bias)
    : max_input(max_input)
    , sum(sum) 
    , bias(bias) {}

  __device__ __forceinline__ __half operator()(__half input) const {
    return __float2half(std::exp(__half2float(input) - max_input) / sum + bias);
  }

  const float max_input;
  const float sum;
  const float bias;
};

inline dim3 SoftMax_getBlockSize(int ILP, uint64_t dim_size) {
  uint64_t block_size = 1;
  uint64_t max_block_size = std::min(dim_size / ILP, static_cast<uint64_t>(max_threads));

  // In the vectorized case we want to trade off allowing more of the buffers to be accessed
  // in a vectorized way against wanting a larger block size to get better utilisation.
  // In general with ILP you can have (ILP-1)/ILP of the buffer accessed vectorised, at the risk
  // of having a very small block size. We choose to keep >= 1/2 of the buffer vectorised while
  // allowing a larger block size.
  if (ILP > 1) {
    max_block_size /= 2;
  }

  while (block_size < (max_block_size)) block_size *= 2;
  // Launch at least a single warp - the kernel assumes that.
  block_size = std::max(block_size, static_cast<uint64_t>(at::cuda::warp_size()));
  return dim3(block_size);
}

template<typename T>
struct Add {
  __device__ __forceinline__ T operator()(T a, T b) const {
    return a + b;
  }
};

template<typename T>
struct Max {
  __device__ __forceinline__ T operator()(T a, T b) const {
    return a < b ? b : a;
  }
};


////////////////////////////////////////////////////////////////////////////////
// Regular kernel (fast when dim_size is large; requires inner_size == 1)
////////////////////////////////////////////////////////////////////////////////


template <typename T, typename AccumT>
struct MaxFloat
{
  __device__ __forceinline__ AccumT operator()(AccumT max, T v) const {
    return ::max(max, (AccumT)v);
  }
};

template <>
struct MaxFloat<__half, float>
{
  __device__ __forceinline__ float operator()(float max, __half v) const {
    return ::max(max, __half2float(v));
  }
};

template<typename T, typename AccumT>
struct AddFloat
{
  __device__ __forceinline__ AccumT operator()(AccumT sum, T v) const {
    return sum + v;
  }
};

template<typename T, typename AccumT>
struct SumExpFloat
{
  __device__ __forceinline__ SumExpFloat(AccumT v)
    : max_k(v) {}

  __device__ __forceinline__ AccumT operator()(AccumT sum, T v) const {
    return sum + std::exp(v - max_k);
  }

  const AccumT max_k;
};

template<>
struct SumExpFloat<__half, float>
{
  __device__ __forceinline__ SumExpFloat(float v)
    : max_k(v) {}

  __device__ __forceinline__ float operator()(float sum, __half v) const {
    return sum + std::exp(__half2float(v) - max_k);
  }

  const float max_k;
};

template <template<typename> class Reduction, typename AccumT>
__device__ __forceinline__ AccumT
blockReduce(AccumT* smem, AccumT val,
            const Reduction<AccumT>& r,
            AccumT defaultVal)
{
  // To avoid RaW races from chaining blockReduce calls together, we need a sync here
  __syncthreads();

  smem[threadIdx.x] = val;

  __syncthreads();

  AccumT warpVal = defaultVal;

  // First warp will perform per-warp reductions for the remaining warps
  uint32_t mask = (((uint64_t)1) << (blockDim.x / C10_WARP_SIZE)) - 1;
  if (threadIdx.x < C10_WARP_SIZE) {
    int lane = threadIdx.x % C10_WARP_SIZE;
    if (lane < blockDim.x / C10_WARP_SIZE) {
#pragma unroll
      for (int i = 0; i < C10_WARP_SIZE; ++i) {
        warpVal = r(warpVal, smem[lane * C10_WARP_SIZE + i]);
      }
#if !defined(USE_ROCM)
      __syncwarp(mask);
#endif
      smem[lane] = warpVal;
    }
  }

  __syncthreads();

  // First thread will perform a reduction of the above per-warp reductions
  AccumT blockVal = defaultVal;

  if (threadIdx.x == 0) {
    for (int i = 0; i < blockDim.x / C10_WARP_SIZE; ++i) {
      blockVal = r(blockVal, smem[i]);
    }
    smem[0] = blockVal;
  }

  // Sync and broadcast
  __syncthreads();
  return smem[0];
}

template <template<typename, typename> class Reduction, int ILP, typename T, typename AccumT>
__device__ __forceinline__ AccumT
ilpReduce(int shift,
          T* data,
          int size,
          const Reduction<T, AccumT>& r,
          AccumT defaultVal)
{
  using LoadT = at::native::memory::aligned_vector<T, ILP>;
  AccumT threadVal = defaultVal;
  int offset = threadIdx.x;

  // shift and do 1
  if(shift > 0){
    data -= shift;
    size += shift;
    if(threadIdx.x >= shift){
      threadVal = r(threadVal, data[offset]);
    }
    size -= blockDim.x;
    data += blockDim.x;
  }
  int last = size % (ILP * blockDim.x);

  T v[ILP];
  LoadT* value = reinterpret_cast<LoadT*>(&v);

  for (; offset * ILP < (size - last); offset += blockDim.x) {
    *value = reinterpret_cast<LoadT*>(data)[offset];

    #pragma unroll
    for (int j = 0; j < ILP; ++j) {
      threadVal = r(threadVal, v[j]);
    }
  }

  offset = size - last + threadIdx.x;
  // Epilogue
  for (; offset < size; offset += blockDim.x)
    threadVal = r(threadVal, data[offset]);

  return threadVal;
}

/**
 * This will apply the Epilogue with vectorized reads & writes when input & output have the same shift
 */
template <int ILP, typename scalar_t, typename accum_t, typename outscalar_t, template<typename, typename, typename> class Epilogue>
__device__ __forceinline__ void
WriteFpropResultsVectorized(
             int size,
             const int shift,
             scalar_t *input,
             outscalar_t *output,
             Epilogue<scalar_t, accum_t, outscalar_t> epilogue) {
  using LoadT = at::native::memory::aligned_vector<scalar_t, ILP>;
  using StoreT = at::native::memory::aligned_vector<outscalar_t, ILP>;

  int offset = threadIdx.x;

  // if unaligned, do one value / thread and move on, guaranteeing aligned reads/writes later
  if (shift > 0) {
    input -= shift;
    output -= shift;
    size += shift;

    if (threadIdx.x >= shift) {
      output[offset] = epilogue(input[offset]);
    }
    size -= blockDim.x;
    input += blockDim.x;
    output += blockDim.x;
  }

  const int last = size % (ILP * blockDim.x);

  scalar_t in_v[ILP];
  LoadT* in_value = reinterpret_cast<LoadT*>(&in_v);

  outscalar_t out_v[ILP];
  StoreT* out_value = reinterpret_cast<StoreT*>(&out_v);

  for (; offset * ILP < (size - last); offset += blockDim.x) {
    *in_value = reinterpret_cast<LoadT*>(input)[offset];

    #pragma unroll
    for (int j = 0; j < ILP; ++j) {
      out_v[j] = epilogue(in_v[j]);
    }

    reinterpret_cast<StoreT*>(output)[offset] = *out_value;
  }

  offset = size - last + threadIdx.x;
  // handle the tail
  for (; offset < size; offset += blockDim.x) {
    output[offset] = epilogue(input[offset]);
  }
}


/**
 * This will apply the Epilogue with non-vectrorized reads & writes for the general case
 */
template <int ILP, typename scalar_t, typename accum_t, typename outscalar_t, template<typename, typename, typename> class Epilogue>
__device__ __forceinline__ void
WriteFpropResults(
             int classes,
             scalar_t *input,
             outscalar_t *output,
             Epilogue<scalar_t, accum_t, outscalar_t> epilogue) {
  int offset = threadIdx.x;

  int last = classes % (ILP * blockDim.x);

  // Main bulk of loop with ILP
  for (; offset < classes - last; offset += blockDim.x * ILP) {
    scalar_t tmp[ILP];

    #pragma unroll
    for (int j = 0; j < ILP; ++j) {
      tmp[j] = input[offset + j * blockDim.x];
    }
    #pragma unroll
    for (int j = 0; j < ILP; ++j) {
      output[offset + j * blockDim.x] = epilogue(tmp[j]);
    }
  }

  // Remainder - no ILP
  for (; offset < classes; offset += blockDim.x) {
    output[offset] = epilogue(input[offset]);
  }
}


template <int ILP, typename scalar_t, typename accscalar_t, typename outscalar_t, template <typename, typename, typename> class Epilogue>
__global__ void
cunn_SoftMaxForward(outscalar_t *output, scalar_t *input, int classes, float bias)
{
  extern __shared__ unsigned char smem[];
  auto sdata = reinterpret_cast<accscalar_t*>(smem);

  using LoadT = at::native::memory::aligned_vector<scalar_t, ILP>;
  using StoreT = at::native::memory::aligned_vector<outscalar_t, ILP>;

  // forward pointers to batch[blockIdx.x]
  // each block handles a sample in the mini-batch
  input += blockIdx.x * classes;
  output += blockIdx.x * classes;

  const int shift = ((uint64_t)input) % ALIGN_BYTES / sizeof(scalar_t);
  const int output_shift = ((uint64_t)output) % ALIGN_BYTES / sizeof(outscalar_t);

  // find the max
  accscalar_t threadMax = ilpReduce<MaxFloat, ILP, scalar_t, accscalar_t>(
      shift, input, classes, MaxFloat<scalar_t, accscalar_t>(), -at::numeric_limits<accscalar_t>::max());
  accscalar_t max_k = blockReduce<Max, accscalar_t>(
      sdata, threadMax, Max<accscalar_t>(), -at::numeric_limits<accscalar_t>::max());

  // reduce all values
  accscalar_t threadExp = ilpReduce<SumExpFloat, ILP, scalar_t, accscalar_t>(
      shift, input, classes, SumExpFloat<scalar_t, accscalar_t>(max_k), static_cast<accscalar_t>(0));
  accscalar_t sumAll = blockReduce<Add, accscalar_t>(
      sdata, threadExp, Add<accscalar_t>(), static_cast<accscalar_t>(0));

  Epilogue<scalar_t, accscalar_t, outscalar_t> epilogue(max_k, sumAll, bias);

  if (shift == output_shift) {
    WriteFpropResultsVectorized<ILP, scalar_t, accscalar_t, outscalar_t, Epilogue>(classes, shift, input, output, epilogue);
  } else {
    WriteFpropResults<ILP, scalar_t, accscalar_t, outscalar_t, Epilogue>(classes, input, output, epilogue);
  }
}

}  // namespace extend
}  // namespace at



torch::Tensor softmax_cuda(torch::Tensor input_, const int64_t dim_, float bias){
  using scalar_t = __half;
  auto input = input_.contiguous();

  auto output = torch::empty_like(input);

  if (input.dim() == 0) input = input.view(1);
  int64_t dim = at::maybe_wrap_dim(dim_, input.dim());
  TORCH_CHECK(dim >=0 && dim < input.dim(), "dim must be non-negative and less than input dimensions");
  int64_t outer_size = 1;
  int64_t dim_size = input.size(dim);

  if (input.numel() > 0) {
    int64_t inner_size = 1;
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    for (int64_t i = 0; i < dim; ++i)
      outer_size *= input.size(i);
    for (int64_t i = dim + 1; i < input.dim(); ++i)
      inner_size *= input.size(i);

    dim3 grid(outer_size);
    using accscalar_t = float;
    constexpr int ILP = sizeof(float4) / sizeof(scalar_t);
    dim3 block = at::extend::SoftMax_getBlockSize(ILP, dim_size);
    at::extend::cunn_SoftMaxForward<ILP, scalar_t, accscalar_t, scalar_t, at::extend::SoftMaxForwardEpilogue>
        <<<grid, block, block.x * sizeof(accscalar_t), stream>>>(
        (scalar_t*) output.data_ptr(), (scalar_t*)input.data_ptr(), dim_size, bias);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  }
  return output;
}