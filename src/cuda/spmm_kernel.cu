#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm_sparse.h"
#include <stdio.h>
#include <vector>
#include "hip/hip_bf16.h"
#include "helper.h"
#include <type_traits>

#include "epilogue/default_epilogue_tensor_op.h"
#include "epilogue/pipelined_transpose_epilogue.h"


// Define the Tile Size in different levels

using ThreadblockShape_bf16 = cutlass::gemm::GemmShape<128, 128, 64>;
using WarpShape_bf16 = cutlass::gemm::GemmShape<64, 64, 64>;
using InstructionShape_bf16 = cutlass::gemm::GemmShape<16, 8, 32>;

// Define MMA & Epilogue
using ThreadblockSwizzle = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;

// DefaultConfigurations for float & bf16
using DefaultConfig = cutlass::gemm::device::DefaultGemmConfiguration<
    cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80, float, float, float, float>;

// Pipeline stages in GEMM
constexpr int NumStages = 3;

// A structure to switch between different configurations
template<typename Element_, typename LayoutB_, bool Trans_>
struct SpMMConfigure{

    static const bool Trans = Trans_;
    using LayoutB = LayoutB_;
    using Element = Element_;

    using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
    Element, 128 / cutlass::sizeof_bits<Element>::value, float, float,
    cutlass::epilogue::thread::ScaleType::OnlyAlphaScaling>;

    using Mma = typename cutlass::gemm::threadblock::DefaultSparseMma<
    Element, cutlass::layout::RowMajor, 128 / cutlass::sizeof_bits<Element>::value,
    Element, LayoutB, 128 / cutlass::sizeof_bits<Element>::value,
    float, cutlass::layout::RowMajor, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,
    ThreadblockShape_bf16, WarpShape_bf16, InstructionShape_bf16, NumStages, cutlass::arch::OpMultiplyAdd>::ThreadblockMma;

    using Epilogue = typename std::conditional<Trans, 
        typename cutlass::epilogue::threadblock::DefaultTransposeEpilogue<
            ThreadblockShape_bf16, WarpShape_bf16, EpilogueOp, Mma>::Epilogue,
        typename cutlass::epilogue::threadblock::DefaultEpilogueTensorOp<
            ThreadblockShape_bf16, typename Mma::Operator, ThreadblockShape_bf16::kK / WarpShape_bf16::kK, EpilogueOp,
            EpilogueOp::kCount>::Epilogue>::type;
    
    union SharedStorage {
        typename Mma::SharedStorage main_loop;
        typename Epilogue::SharedStorage epilogue;
    }; 
};


template<typename Element, typename _Mma, typename _SharedStorage, typename _Epilogue>
__device__ void cutlassSpmmKernel_16_(
    cutlass::gemm::GemmCoord problem_size,
    cutlass::gemm::GemmCoord grid_tiled_shape,
    typename _Mma::IteratorA::Params params_A,
    Element* __restrict__ ptr_A,
    typename _Mma::IteratorB::Params params_B,
    Element* __restrict__ ptr_B,
    typename _Epilogue::OutputTileIterator::Params params_D,
    Element* __restrict__ ptr_D,
    typename _Mma::IteratorE::Params params_E,
    typename _Mma::ElementE* __restrict__ ptr_E,
    typename _Epilogue::OutputOp::Params output_op_,
    int gemm_k_size)
{
    extern __shared__ int SharedStorageBase[];

    _SharedStorage& shared_storage = *reinterpret_cast<_SharedStorage *>(SharedStorageBase);

    ThreadblockSwizzle threadblock_swizzle;

    cutlass::gemm::GemmCoord threadblock_tile_offset=threadblock_swizzle.get_tile_offset(grid_tiled_shape);

    // Early exit if CTA is out of range
    if (grid_tiled_shape.m() <= threadblock_tile_offset.m() ||
        grid_tiled_shape.n() <= threadblock_tile_offset.n())
    {
        return;
    }

    // Compute initial location in logical coordinates
    cutlass::MatrixCoord tb_offset_A{
        threadblock_tile_offset.m() * _Mma::Shape::kM,
        threadblock_tile_offset.k() * gemm_k_size / _Mma::kSparse
    };

    cutlass::MatrixCoord tb_offset_B{
        threadblock_tile_offset.k() * gemm_k_size,
        threadblock_tile_offset.n() * _Mma::Shape::kN
    };

    cutlass::MatrixCoord tb_offset_E{
        threadblock_tile_offset.m() * _Mma::Shape::kM,
        threadblock_tile_offset.k() * gemm_k_size / _Mma::kSparse
    };

    // Problem size
    int problem_size_k = min(problem_size.k(), (threadblock_tile_offset.k() + 1) * gemm_k_size);

    int gemm_k_iterations = (problem_size_k - tb_offset_B.row() + _Mma::Shape::kK - 1) / _Mma::Shape::kK;

    // Compute position within threadblock
    int thread_idx = threadIdx.x;

    // Construct iterators to A, B, and E operands
    typename _Mma::IteratorA iterator_A(
        params_A,
        //ref_A.data(),
        ptr_A,
        {problem_size.m(), problem_size_k / _Mma::kSparse},
        thread_idx,
        tb_offset_A
    );

    typename _Mma::IteratorB iterator_B(
        params_B,
        //ref_B.data(),
        ptr_B,
        {problem_size_k, problem_size.n()},
        thread_idx,
        tb_offset_B
    );

    typename _Mma::IteratorE iterator_E(
        params_E,
        // ref_E.data(),
        ptr_E,
        {problem_size.m(),
        problem_size_k / _Mma::kSparse / _Mma::kElementsPerElementE},
        thread_idx,
        tb_offset_E
    );

    // Broadcast the warp_id computed by lane 0 to ensure dependent code
    // is compuled as warp-uniform
    int warp_idx = __shfl_sync(0xffffffff, threadIdx.x / 32, 0);
    int lane_idx = threadIdx.x % 32;

    //
    //  Main loop
    //

    // Construct thread-scoped matrix multiply
    _Mma mma(shared_storage.main_loop, thread_idx, warp_idx, lane_idx);

    typename _Mma::FragmentC accumulators;

    accumulators.clear();

    if (gemm_k_iterations > 0){
        mma(gemm_k_iterations, accumulators, iterator_A, iterator_B, iterator_E, accumulators);
    }

    //
    //  Epilogue
    //

    typename _Epilogue::OutputOp output_op(output_op_);

    threadblock_tile_offset = threadblock_swizzle.get_tile_offset(grid_tiled_shape);

    // (blockIdx.x * TileM, blockIdx.y * TileN)
    cutlass::MatrixCoord threadblock_offset(
        threadblock_tile_offset.m() * _Mma::Shape::kM,
        threadblock_tile_offset.n() * _Mma::Shape::kN
    );

    
    typename _Epilogue::OutputTileIterator iterator_D(
        params_D,
        ptr_D,
        problem_size.mn(),
        thread_idx,
        threadblock_offset
    );

    
    _Epilogue epilogue(
        shared_storage.epilogue,
        thread_idx,
        warp_idx,
        lane_idx
    );

    epilogue(output_op, iterator_D, accumulators, iterator_D);
}


template<typename Element, typename Config>
__global__ void cutlassSpmmKernel_16(
    cutlass::gemm::GemmCoord problem_size,
    cutlass::gemm::GemmCoord grid_tiled_shape,
    typename Config::Mma::IteratorA::Params params_A,
    Element* __restrict__ ptr_A,
    typename Config::Mma::IteratorB::Params params_B,
    Element* __restrict__ ptr_B,
    typename Config::Epilogue::OutputTileIterator::Params params_D,
    Element* __restrict__ ptr_D,
    typename Config::Mma::IteratorE::Params params_E,
    typename Config::Mma::ElementE* __restrict__ ptr_E,
    typename Config::Epilogue::OutputOp::Params output_op_,
    int gemm_k_size)
{
    cutlassSpmmKernel_16_<Element, Config::Mma, Config::SharedStorage, Config::Epilogue>(
        problem_size, grid_tiled_shape,
        params_A, ptr_A, params_B, ptr_B,
        params_D, ptr_D, params_E, ptr_E,
        output_op_, gemm_k_size
    );
}


template<typename Config>
torch::Tensor spmm_cuda(
    torch::Tensor tensor_a,
    torch::Tensor tensor_b,
    torch::Tensor tensor_e)
{
    int m, n, k;
    m = tensor_a.size(0);
    if (std::is_same<typename Config::LayoutB, cutlass::layout::RowMajor>::value){
        n = tensor_b.size(1); 
        k = tensor_b.size(0);   
    } else {
        n = tensor_b.size(0);
        k = tensor_b.size(1);
    }

    auto options_val = torch::TensorOptions().dtype(tensor_a.dtype()).device(tensor_b.device());
    torch::Tensor output_matrix;
    if (Config::Trans){
        output_matrix = torch::empty({n, m}, options_val);
    } else {
        output_matrix = torch::empty({m, n}, options_val);
    }

    // Create a tuple of problem size for matrix multiplication
    cutlass::gemm::GemmCoord problem_size(m, n, k);

    auto layout_a = cutlass::layout::RowMajor::packed(cutlass::make_Coord(problem_size.m(), problem_size.k() / 2));
    auto layout_b = Config::LayoutB::packed(problem_size.kn());
    auto layout_e = Config::Mma::LayoutE::packed(cutlass::make_Coord(problem_size.m(), problem_size.k()/Config::Mma::kSparse / Config::Mma::kElementsPerElementE));
    auto layout_d = cutlass::layout::RowMajor::packed(problem_size.mn());

    typename Config::Element alpha = typename Config::Element(1.0);
    typename Config::Element beta = typename Config::Element(0.0);
    
    ThreadblockSwizzle threadblock_swizzle;

    cutlass::gemm::GemmCoord grid_tiled_shape = threadblock_swizzle.get_tiled_shape(
        problem_size,
        {ThreadblockShape_bf16::kM, ThreadblockShape_bf16::kN, ThreadblockShape_bf16::kK},
        1
    );

    dim3 grid = threadblock_swizzle.get_grid_shape(grid_tiled_shape);
    dim3 block(Config::Mma::WarpCount::kCount * 32, 1, 1);

    int smem_size = int(sizeof(typename Config::SharedStorage));

    hipFuncSetAttribute(reinterpret_cast<const void*>(cutlassSpmmKernel_16<typename Config::Element), Config>, hipFuncAttributeMaxDynamicSharedMemorySize, smem_size);
    hipFuncSetAttribute(reinterpret_cast<const void*>(cutlassSpmmKernel_16<typename Config::Element), Config>, hipFuncAttributePreferredSharedMemoryCarveout, 100);

    int gemm_k_size = ((problem_size.k() + Config::Mma::Shape::kK - 1) / Config::Mma::Shape::kK) * Config::Mma::Shape::kK;

    cutlassSpmmKernel_16<typename Config::Element, Config><<<grid, block, smem_size>>>(
        problem_size, grid_tiled_shape, 
        layout_a, (typename Config::Element*)tensor_a.data_ptr(),
        layout_b, (typename Config::Element*)tensor_b.data_ptr(),
        layout_d, (typename Config::Element*)output_matrix.data_ptr(),
        layout_e, (typename Config::Mma::ElementE*)tensor_e.data_ptr(),
        {alpha, beta}, gemm_k_size);

    return output_matrix;
}


torch::Tensor spmmv2_bf16_nnn_cuda(
    torch::Tensor tensor_a,
    torch::Tensor tensor_b,
    torch::Tensor tensor_e_reordered)
{
    using Config = SpMMConfigure<cutlass::bfloat16_t, cutlass::layout::RowMajor, false>;
    return spmm_cuda<Config>(tensor_a, tensor_b, tensor_e_reordered);
}


torch::Tensor spmmv2_bf16_ntn_cuda(
    torch::Tensor tensor_a,
    torch::Tensor tensor_b,
    torch::Tensor tensor_e_reordered)
{
    using Config = SpMMConfigure<cutlass::bfloat16_t, cutlass::layout::ColumnMajor, false>;
    return spmm_cuda<Config>(tensor_a, tensor_b, tensor_e_reordered);
}


torch::Tensor spmmv2_bf16_ntt_cuda(
    torch::Tensor tensor_a,
    torch::Tensor tensor_b,
    torch::Tensor tensor_e_reordered)
{
    using Config = SpMMConfigure<cutlass::bfloat16_t, cutlass::layout::ColumnMajor, true>;
    return spmm_cuda<Config>(tensor_a, tensor_b, tensor_e_reordered);
}


torch::Tensor spmmv2_f16_nnn_cuda(
    torch::Tensor tensor_a,
    torch::Tensor tensor_b,
    torch::Tensor tensor_e_reordered)
{
    using Config = SpMMConfigure<cutlass::half_t, cutlass::layout::RowMajor, false>;
    return spmm_cuda<Config>(tensor_a, tensor_b, tensor_e_reordered);
}


torch::Tensor spmmv2_f16_ntn_cuda(
    torch::Tensor tensor_a,
    torch::Tensor tensor_b,
    torch::Tensor tensor_e_reordered)
{
    using Config = SpMMConfigure<cutlass::half_t, cutlass::layout::ColumnMajor, false>;
    return spmm_cuda<Config>(tensor_a, tensor_b, tensor_e_reordered);
}


torch::Tensor spmmv2_f16_ntt_cuda(
    torch::Tensor tensor_a,
    torch::Tensor tensor_b,
    torch::Tensor tensor_e_reordered)
{
    using Config = SpMMConfigure<cutlass::half_t, cutlass::layout::ColumnMajor, true>;
    return spmm_cuda<Config>(tensor_a, tensor_b, tensor_e_reordered);
}