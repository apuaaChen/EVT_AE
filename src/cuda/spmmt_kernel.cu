#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm_sparse.h"
#include <stdio.h>
#include <vector>
#include "hip/hip_bf16.h"
#include "helper.h"
#include "mma/default_mma.h"
#include <type_traits>

#include "spmmt/default_sparse_mma_trans.h"
#include "epilogue/pipelined_transpose_epilogue.h"

// Define the Tile Size in different levels

using ThreadblockShape_16 = cutlass::gemm::GemmShape<128, 256, 64>;
using WarpShape_16 = cutlass::gemm::GemmShape<32, 128, 64>;
using InstructionShape_16 = cutlass::gemm::GemmShape<16, 8, 32>;

// using ThreadblockShape_f16 = cutlass::gemm::GemmShape<128, 256, 64>;
// using WarpShape_f16 = cutlass::gemm::GemmShape<32, 128, 64>;
// using InstructionShape_f16 = cutlass::gemm::GemmShape<16, 8, 32>;

// Define MMA & Epilogue
using ThreadblockSwizzle = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;

// DefaultConfigurations for float & bf16
using DefaultConfig = cutlass::gemm::device::DefaultGemmConfiguration<
    cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80, float, float, float, float>;

// Pipeline stages in GEMM
constexpr int NumStages = 3;


// A structure to switch between different configurations
template<typename Element_, bool Trans_>
struct SpMMTConfigure{
    static const bool Trans = Trans_;
    using Element = Element_;

    using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
        Element, 128 / cutlass::sizeof_bits<Element>::value, float, float,
        cutlass::epilogue::thread::ScaleType::OnlyAlphaScaling>;  
    
    using Mma = typename cutlass::gemm::threadblock::DefaultSparseMmaTrans<
        Element, cutlass::layout::RowMajor, 128 / cutlass::sizeof_bits<Element>::value,
        Element, cutlass::layout::ColumnMajor, 128 / cutlass::sizeof_bits<Element>::value,
        float, cutlass::layout::RowMajor, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,
        ThreadblockShape_16, WarpShape_16, InstructionShape_16, NumStages, cutlass::arch::OpMultiplyAdd>::ThreadblockMma;
    
    using Epilogue = typename std::conditional<Trans,
        typename cutlass::epilogue::threadblock::DefaultTransposeEpilogue<
            ThreadblockShape_16, WarpShape_16, EpilogueOp, Mma>::Epilogue,
        typename cutlass::epilogue::threadblock::DefaultEpilogueTensorOp<
            ThreadblockShape_16, typename Mma::Operator, ThreadblockShape_16::kK / WarpShape_16::kK, EpilogueOp,
            EpilogueOp::kCount>::Epilogue>::type;
    
    union SharedStorage {
        typename Mma::SharedStorage main_loop;
        typename Epilogue::SharedStorage epilogue;
    };
};


template<typename Element, typename _Mma, typename _SharedStorage, typename _Epilogue>
__global__ void cutlassSpmmTKernel_16(
    cutlass::gemm::GemmCoord problem_size,
    cutlass::gemm::GemmCoord grid_tiled_shape,
    typename _Mma::IteratorA::Params params_A,
    Element* __restrict__ ptr_A,
    typename _Mma::IteratorB::Params params_B,
    Element* __restrict__ ptr_B,
    typename _Epilogue::OutputTileIterator::Params params_D,
    Element* __restrict__ ptr_D,
    typename _Mma::IteratorE::Params params_E,
    typename _Mma::ElementE* __restrict__ ptr_E,
    typename _Epilogue::OutputOp::Params output_op_,
    int gemm_k_size)
{
    extern __shared__ int SharedStorageBase[];

    _SharedStorage& shared_storage = *reinterpret_cast<_SharedStorage *>(SharedStorageBase);

    ThreadblockSwizzle threadblock_swizzle;

    cutlass::gemm::GemmCoord threadblock_tile_offset=threadblock_swizzle.get_tile_offset(grid_tiled_shape);

    // Early exit if CTA is out of range
    if (grid_tiled_shape.m() <= threadblock_tile_offset.m() ||
        grid_tiled_shape.n() <= threadblock_tile_offset.n())
    {
        return;
    }

    // Compute initial location in logical coordinates
    cutlass::MatrixCoord tb_offset_A{
        threadblock_tile_offset.k() * gemm_k_size,
        threadblock_tile_offset.m() * _Mma::Shape::kM / _Mma::kSparse
    };

    cutlass::MatrixCoord tb_offset_B{
        threadblock_tile_offset.k() * gemm_k_size,
        threadblock_tile_offset.n() * _Mma::Shape::kN
    };

    cutlass::MatrixCoord tb_offset_E{
        threadblock_tile_offset.k() * gemm_k_size,
        threadblock_tile_offset.m() * _Mma::Shape::kM / _Mma::kSparse / _Mma::kElementsPerElementE
    };

    // Problem size
    int problem_size_k = min(problem_size.k(), (threadblock_tile_offset.k() + 1) * gemm_k_size);

    int gemm_k_iterations = (problem_size_k - tb_offset_B.row() + _Mma::Shape::kK - 1) / _Mma::Shape::kK;

    // Compute position within threadblock
    int thread_idx = threadIdx.x;

    // Construct iterators to A, B, and E operands
    typename _Mma::IteratorA iterator_A(
        params_A,
        //ref_A.data(),
        ptr_A,
        {problem_size_k, problem_size.m() / _Mma::kSparse},
        thread_idx,
        tb_offset_A
    );

    typename _Mma::IteratorB iterator_B(
        params_B,
        //ref_B.data(),
        ptr_B,
        {problem_size_k, problem_size.n()},
        thread_idx,
        tb_offset_B
    );

    typename _Mma::IteratorE iterator_E(
        params_E,
        // ref_E.data(),
        ptr_E,
        {problem_size_k,
        problem_size.m() / _Mma::kSparse / _Mma::kElementsPerElementE},
        thread_idx,
        tb_offset_E
    );

    // Broadcast the warp_id computed by lane 0 to ensure dependent code
    // is compuled as warp-uniform
    int warp_idx = __shfl_sync(0xffffffff, threadIdx.x / 32, 0);
    int lane_idx = threadIdx.x % 32;

    //
    //  Main loop
    //

    // Construct thread-scoped matrix multiply
    _Mma mma(shared_storage.main_loop, thread_idx, warp_idx, lane_idx);

    typename _Mma::FragmentC accumulators;

    accumulators.clear();

    if (gemm_k_iterations > 0){
        mma(gemm_k_iterations, accumulators, iterator_A, iterator_B, iterator_E, accumulators);
    }

    //
    //  Epilogue
    //

    typename _Epilogue::OutputOp output_op(output_op_);

    threadblock_tile_offset = threadblock_swizzle.get_tile_offset(grid_tiled_shape);

    // (blockIdx.x * TileM, blockIdx.y * TileN)
    cutlass::MatrixCoord threadblock_offset(
        threadblock_tile_offset.m() * _Mma::Shape::kM,
        threadblock_tile_offset.n() * _Mma::Shape::kN
    );

    int block_idx = threadblock_tile_offset.m() + threadblock_tile_offset.n() * grid_tiled_shape.m();
    
    typename _Epilogue::OutputTileIterator iterator_D(
        params_D,
        ptr_D,
        problem_size.mn(),
        thread_idx,
        threadblock_offset
    );

    
    _Epilogue epilogue(
        shared_storage.epilogue,
        thread_idx,
        warp_idx,
        lane_idx
    );

    epilogue(output_op, iterator_D, accumulators, iterator_D);
}


template<typename Config>
torch::Tensor spmmt_cuda(
    torch::Tensor tensor_a,
    torch::Tensor tensor_b,
    torch::Tensor tensor_e)
{
    const int m = tensor_a.size(1) * 2;
    const int n = tensor_b.size(0);
    const int k = tensor_b.size(1);

    auto options_val = torch::TensorOptions().dtype(tensor_a.dtype()).device(tensor_b.device());
    torch::Tensor output_matrix;
    if (Config::Trans){
        output_matrix = torch::empty({n, m}, options_val);
    } else {
        output_matrix = torch::empty({m, n}, options_val);
    }

    // Create a tuple of problem size for matrix multiplication
    cutlass::gemm::GemmCoord problem_size(m, n, k);

    auto layout_a = cutlass::layout::RowMajor::packed(cutlass::make_Coord(problem_size.k(), problem_size.m()/2));
    auto layout_b = cutlass::layout::ColumnMajor::packed(problem_size.kn());
    auto layout_e = Config::Mma::LayoutE::packed(cutlass::make_Coord(problem_size.k(), problem_size.m()/Config::Mma::kSparse / Config::Mma::kElementsPerElementE));
    auto layout_d = cutlass::layout::RowMajor::packed(problem_size.mn());

    typename Config::Element alpha = typename Config::Element(1.0);
    typename Config::Element beta = typename Config::Element(0.0);
    
    ThreadblockSwizzle threadblock_swizzle;

    cutlass::gemm::GemmCoord grid_tiled_shape = threadblock_swizzle.get_tiled_shape(
        problem_size,
        {ThreadblockShape_16::kM, ThreadblockShape_16::kN, ThreadblockShape_16::kK},
        1
    );

    dim3 grid = threadblock_swizzle.get_grid_shape(grid_tiled_shape);
    dim3 block(Config::Mma::WarpCount::kCount * 32, 1, 1);

    int smem_size = int(sizeof(typename Config::SharedStorage));

    hipFuncSetAttribute(reinterpret_cast<const void*>(cutlassSpmmTKernel_16<typename Config::Element), typename Config::Mma, typename Config::SharedStorage, typename Config::Epilogue>, hipFuncAttributeMaxDynamicSharedMemorySize, smem_size);
    hipFuncSetAttribute(reinterpret_cast<const void*>(cutlassSpmmTKernel_16<typename Config::Element), typename Config::Mma, typename Config::SharedStorage, typename Config::Epilogue>, hipFuncAttributePreferredSharedMemoryCarveout, 100);

    int gemm_k_size = ((problem_size.k() + Config::Mma::Shape::kK - 1) / Config::Mma::Shape::kK) * Config::Mma::Shape::kK;

    cutlassSpmmTKernel_16<typename Config::Element, typename Config::Mma, typename Config::SharedStorage, typename Config::Epilogue><<<grid, block, smem_size>>>(
        problem_size, grid_tiled_shape, 
        layout_a, (typename Config::Element*)tensor_a.data_ptr(),
        layout_b, (typename Config::Element*)tensor_b.data_ptr(),
        layout_d, (typename Config::Element*)output_matrix.data_ptr(),
        layout_e, (typename Config::Mma::ElementE*)tensor_e.data_ptr(),
        {alpha, beta}, gemm_k_size);

    return output_matrix;
}


torch::Tensor spmmt_bf16_ntn_cuda(
    torch::Tensor tensor_a,
    torch::Tensor tensor_b,
    torch::Tensor tensor_e_reordered)
{
    using Config = SpMMTConfigure<cutlass::bfloat16_t, false>;
    return spmmt_cuda<Config>(tensor_a, tensor_b, tensor_e_reordered);
}

torch::Tensor spmmt_bf16_ntt_cuda(
    torch::Tensor tensor_a,
    torch::Tensor tensor_b,
    torch::Tensor tensor_e_reordered)
{
    using Config = SpMMTConfigure<cutlass::bfloat16_t, true>;
    return spmmt_cuda<Config>(tensor_a, tensor_b, tensor_e_reordered);
}

torch::Tensor spmmt_f16_ntn_cuda(
    torch::Tensor tensor_a,
    torch::Tensor tensor_b,
    torch::Tensor tensor_e_reordered)
{
    using Config = SpMMTConfigure<cutlass::half_t, false>;
    return spmmt_cuda<Config>(tensor_a, tensor_b, tensor_e_reordered);
}

torch::Tensor spmmt_f16_ntt_cuda(
    torch::Tensor tensor_a,
    torch::Tensor tensor_b,
    torch::Tensor tensor_e_reordered)
{
    using Config = SpMMTConfigure<cutlass::half_t, true>;
    return spmmt_cuda<Config>(tensor_a, tensor_b, tensor_e_reordered);
}
